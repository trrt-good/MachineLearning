#include "hip/hip_runtime.h"
#include "neural_net_gpu.h"

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
 * @brief npl = NODES PER LAYER. this is the central array
 * which the neural network is built from.
 * Index 0 of the array is how many input neurons there are.
 * The last index is how many output neurons there are.
 * There can be as many numbers inbetween as desired and each
 * symbolizes the number of hidden neurons there are in the layer
 * corresponding to it's index.
 */
const int npl[LAYERS + 1] = {INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZES, OUTPUT_LAYER_SIZE};

// function declairations
void multiply_MtVreplace(float **matrixVals, int rows, int cols, float *vectorVals, float *buffer);

void nnet_subtract_gradients(float *d_weights[LAYERS], float *d_weight_gradients[LAYERS], float *d_biases[LAYERS], float *d_bias_gradients[LAYERS], float learn_rate, int batch_size);

float nnet_cost_function_MSE(float *outputs, float *expected);
float nnet_cost_function_CCE(float *outputs, float *expected);

void nnet_layer_function_dense_deriv_weights_and_biases(float *current_chain_deriv, int current_layer, float *activation, float **weight_gradients[LAYERS], float *bias_gradients[LAYERS]);
void nnet_layer_function_dense_deriv_activations(NeuralNet* nnet, float* current_chain_deriv, float* math_buffer, int current_layer);
void nnet_activation_function_deriv_relu(float* current_chain_deriv, int current_layer, float* activation);
void nnet_activation_function_deriv_sigmoid(float* current_chain_deriv, int current_layer, float* activation);
void nnet_cost_function_deriv_MSE(float* destination, float *activations, float* training_output);

void nnet_backprop(NeuralNet *nnet, float *activations[LAYERS], float **weight_gradients[LAYERS], float *bias_gradients[LAYERS], float *current_chain_deriv, float *math_buffer, float *training_input, float *training_output);

float sigmoid(float n)
{
    return (1 / (1 + powf(2.71828183F, -n)));
}

float relu(float n)
{
    return n*(n>0);
}

NeuralNet *nnet_init(float init_min, float init_max)
{
    NeuralNet *new_network = (NeuralNet *)malloc(sizeof(NeuralNet));
    int i;
    for (i = 0; i < LAYERS; i++)
    {
        new_network->weights[i] = laa_allocRandMatrix(npl[i + 1], npl[i], init_min, init_max);
        new_network->biases[i] = laa_allocRandVector(npl[i + 1], init_min, init_max);
    }
    return new_network;
}

void nnet_free(NeuralNet *nnet)
{
    int i;
    for (i = 0; i < LAYERS; i++)
    {
        laa_freeMatrix(nnet->weights[i], npl[i + 1]);
        laa_freeVector(nnet->biases[i]);
    }
    free(nnet);
}

void nnet_reset_network(NeuralNet *nnet)
{
    srand(rand() % 0xffffffff);
    int i;
    for (i = 0; i < LAYERS; i++)
    {
        laa_setMatrixToRand(nnet->weights[i], npl[i + 1], npl[i]);
        laa_setVectorToRand(nnet->biases[i], npl[i + 1]);
    }
}

void nnet_free_gpu_wba(float *d_weights[LAYERS], float *d_weight_gradients[LAYERS], float *d_biases[LAYERS], float *d_bias_gradients[LAYERS], float *d_activations[LAYERS])
{
    for (int i = 0; i < LAYERS; i ++)
    {
        cudaErrorCheck(hipFree(d_activations[i]));
        cudaErrorCheck(hipFree(d_biases[i]));
        cudaErrorCheck(hipFree(d_bias_gradients[i]));
        cudaErrorCheck(hipFree(d_weights[i]));
        cudaErrorCheck(hipFree(d_weight_gradients[i]));
    }
}

void nnet_alloc_gpu_wba(float *d_weights[LAYERS], float *d_weight_gradients[LAYERS], float *d_biases[LAYERS], float *d_bias_gradients[LAYERS], float *d_activations[LAYERS])
{
    for (int i = 0; i < LAYERS; i ++)
    {
        int layer_size = sizeof(float) * npl[i+1];
        cudaErrorCheck(hipMalloc((void **)&d_weights[i], layer_size * npl[i]));
        cudaErrorCheck(hipMalloc((void **)&d_weight_gradients[i], layer_size * npl[i]));
        cudaErrorCheck(hipMalloc((void **)&d_biases[i], layer_size));
        cudaErrorCheck(hipMalloc((void **)&d_bias_gradients[i], layer_size));
        cudaErrorCheck(hipMalloc((void **)&d_activations[i], layer_size));
    }
}

void nnet_alloc_gpu_data(float *d_training_inputs, float *d_training_outputs, int num_examples)
{
    cudaErrorCheck(hipMalloc((void **)&d_training_inputs, INPUT_LAYER_SIZE * num_examples * sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_training_outputs, OUTPUT_LAYER_SIZE * num_examples * sizeof(float)));
}

void nnet_free_gpu_data(float *d_training_inputs, float *d_training_outputs)
{
    cudaErrorCheck(hipFree(d_training_inputs));
    cudaErrorCheck(hipFree(d_training_outputs));
}

float nnet_cost_function_MSE(float *outputs, float *expected)
{
    float sum = 0;
    for (int i = 0; i < npl[LAYERS]; i++)
        sum += (outputs[i] - expected[i]) * (outputs[i] - expected[i]);
    return sum;
}

float nnet_cost_function_CCE(float *outputs, float *expected) 
{
    float total = 0.0;
    for (int i = 0; i < npl[LAYERS]; i++)
        total += expected[i] * log(outputs[i]);
    return -total;
}

//cpu method. Predictions and outputs should be on cpu RAM
float nnet_total_cost(float **correct_outputs, float ** predictions, int num_data_points)
{
    float sum = 0;
    for (int i = 0; i < num_data_points; i++)
    {
        sum += nnet_cost_function_MSE(correct_outputs[i], predictions[i]) / num_data_points;
    }
    return sum;
}

//computes the activation of each neuron in a layer in parallel
//all parameters must be pointing to gpu memory
__global__ void nnet_kernel_layer_function_dense_relu(float *d_weights, int rows, int columns, float *d_activations, float *d_bias, float *d_destination)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    float temp_sum = 0.0;

    if (row < rows)
    {
        for (int i = 0; i < columns; i ++)
        {
            temp_sum += d_weights[row * columns + i] * d_activations[i];
        }
        temp_sum += d_bias[row];
        d_destination[row] = temp_sum * (temp_sum > 0);
    }
}

//inputs array must be pointing to gpu memory
void nnet_feed_forward(float *d_inputs, float *d_weights[LAYERS], float *d_biases[LAYERS], float *d_activations[LAYERS])
{
    int i;
    //hidden layers:
    dim3 grid_size((MAX_LAYER_SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE);
    dim3 block_size(BLOCK_SIZE);
    nnet_kernel_layer_function_dense_relu<<<grid_size, block_size>>>(d_weights[0], npl[1], npl[0], d_inputs, d_biases[0], d_activations[0]);
    for (i = 1; i < LAYERS-1; i++)
        nnet_kernel_layer_function_dense_relu<<<grid_size, block_size>>>(d_weights[i], npl[i + 1], npl[i], d_activations[i - 1], d_biases[i], d_activations[i]);

    //last/output layer:
    nnet_kernel_layer_function_dense_relu<<<grid_size, block_size>>>(d_weights[i], npl[i + 1], npl[i], d_activations[i - 1], d_biases[i], d_activations[i]);
}

__global__ void nnnet_kernel_layer_function_dense_deriv_weights_and_biases(float *d_current_chain_deriv, int current_layer, float *d_activation, float *d_weight_gradient, float *d_bias_gradient)
{
    const int d_npl[LAYERS + 1] = {INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZES, OUTPUT_LAYER_SIZE};
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j;
    if (i < d_npl[current_layer+1])
    {
        for (j = 0; j < d_npl[current_layer]; j ++)
        {
            d_weight_gradient[i*d_npl[current_layer] + j] += d_current_chain_deriv[i]*d_activation[j];
        }
        d_bias_gradient[i] += d_current_chain_deriv[i];
    }
}

// calculates the gradient of the weights and biases of the current layer by multiplying their derivatives with respect to the current layer by the 
// current chain-rule product. 
void nnet_layer_function_dense_deriv_weights_and_biases(float *current_chain_deriv, int current_layer, float *activation, float **weight_gradients[LAYERS], float *bias_gradients[LAYERS])
{
    // computes current_chain_deriv * activations' and adds result into weight gradient
    for (int i = 0; i < npl[current_layer+1]; i ++)
    {
        for (int j = 0; j < npl[current_layer]; j++)
        {
            weight_gradients[current_layer][i][j] += current_chain_deriv[i]*activation[j];
        }
        bias_gradients[current_layer][i] += current_chain_deriv[i];
    }
}

__global__ void nnet_kernel_layer_function_dense_deriv_activations(float *d_weights, int current_layer, float *d_current_chain_deriv)
{
    const int d_npl[LAYERS + 1] = {INPUT_LAYER_SIZE, HIDDEN_LAYER_SIZES, OUTPUT_LAYER_SIZE};
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ int temp[MAX_LAYER_SIZE];
    float sum = 0;
    int j;

    if (i < d_npl[current_layer + 1])
        temp[i] = d_current_chain_deriv[i];
    __syncthreads();
    if (i < d_npl[current_layer])
    {
        for (j = 0; j < d_npl[current_layer + 1]; j++)
        {
            sum += d_weights[j * d_npl[current_layer] + i] * temp[i];
        }
        d_current_chain_deriv[i] = sum;
    }
}

//untransposed rows and cols
//afterwords the vectorvals should have length of cols
void multiply_MtVreplace(float **matrixVals, int rows, int cols, float *vectorVals, float *buffer)
{
    int i, j;
    float sum = 0;

    for (i = 0; i < rows; i ++)
    {
        buffer[i] = vectorVals[i];
    }

    for (i = 0; i < cols; i++)
    {
        sum = 0;
        for (j = 0; j < rows; j++)
        {
            sum += matrixVals[j][i]*buffer[j];
        }
        vectorVals[i] = sum;
    }
}

// calculates the derivative of the current layer's activations with respect to the last layer's, and multiplies that by the 
// current chain-rule product to update it. 
void nnet_layer_function_dense_deriv_activations(NeuralNet* nnet, float* current_chain_deriv, float* math_buffer, int current_layer)
{
    multiply_MtVreplace(nnet->weights[current_layer], npl[current_layer+1], npl[current_layer], current_chain_deriv, math_buffer);
} 

//layer_size should be equal to npl[current_layer+1]
__global__ void nnet_kernel_activation_function_deriv_relu(float *d_current_chain_deriv, int layer_size, float *d_activation)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < layer_size)
        d_current_chain_deriv[i] *= (d_activation[i] > 0);
}

void nnet_activation_function_deriv_relu(float* current_chain_deriv, int current_layer, float* activation)
{
    for (int i = 0; i < npl[current_layer+1]; i ++)
        current_chain_deriv[i] *= (activation[i] > 0);
}

void nnet_activation_function_deriv_sigmoid(float* current_chain_deriv, int current_layer, float* activation)
{
    float temp;
    for (int i = 0; i < npl[current_layer+1]; i ++)
    {
        temp = sigmoid(activation[i]);
        current_chain_deriv[i] *= temp*(1-temp);
    }
}

//layer_size is equal to npl[LAYERS]
__global__ void nnet_kernel_cost_function_deriv_MSE(float *d_destination, float *d_activations, float *d_training_output, int layer_size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < layer_size)
        d_destination[i] = d_activations[i] - d_training_output[i];
}

//means squared error derivative. Derivative of the cost function with respect to the output of the nnet (activation at last layer)
//places the result in the destination matrix. Column vector with values ai-yi
void nnet_cost_function_deriv_MSE(float* destination, float *activations, float* training_output)
{
    for (int i = 0; i < npl[LAYERS]; i ++)
        destination[i] = activations[i] - training_output[i];
}

// outputs the derivative of the categorical cross entropy loss function with respect to the activations 
// (predictions from the neural net), to the destination array, given the expected predictions (training_output)
void nnet_cost_function_deriv_CCE_softmax(float* destination, float *activations, float* training_output)
{
    for(int i = 0; i < npl[LAYERS]; i++)
        destination[i] = (activations[i] - training_output[i]);
}

void nnet_backprop(float *d_weights[LAYERS], float *d_biases[LAYERS], float *d_activations[LAYERS], float *d_weight_gradients[LAYERS], float *d_bias_gradients[LAYERS], float *d_current_chain_deriv, float *d_training_input, float *d_training_output)
{
    int layer = LAYERS-1; 

    nnet_feed_forward(d_training_input, d_weights, d_biases, d_activations);

    dim3 grid_size((MAX_LAYER_SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE);
    dim3 block_size(BLOCK_SIZE);
    
    //set the current chain rule derivative value to the derivative of the cost function with respect to the last activation.
    nnet_kernel_cost_function_deriv_MSE<<<grid_size, block_size>>>(d_current_chain_deriv, d_activations[LAYERS-1], d_training_output, npl[LAYERS]);

    for (layer = LAYERS - 1; layer > 0; layer--)
    {
        //applies derivative of activation function (relu, sigmoid, etc) to the current_chain_deriv, in accordance to the chain rule
        nnet_kernel_activation_function_deriv_relu<<<grid_size, block_size>>>(d_current_chain_deriv, npl[layer+1], d_activations[layer]);

        //updates the weight and bias gradients based on the current_chain_deriv.
        nnnet_kernel_layer_function_dense_deriv_weights_and_biases<<<grid_size, block_size>>>(d_current_chain_deriv, layer, d_activations[layer-1], d_weight_gradients[layer], d_bias_gradients[layer]);

        //updates the current_chain_deriv matrix for the next layer function derivative.
        nnet_kernel_layer_function_dense_deriv_activations<<<grid_size, block_size>>>(d_weights[layer], layer, d_current_chain_deriv);
    }
    //the first layer
    nnet_kernel_activation_function_deriv_relu<<<grid_size, block_size>>>(d_current_chain_deriv, npl[layer+1], d_activations[layer]);
    nnnet_kernel_layer_function_dense_deriv_weights_and_biases<<<grid_size, block_size>>>(d_current_chain_deriv, layer, d_training_input, d_weight_gradients[layer], d_bias_gradients[layer]);
    nnet_kernel_layer_function_dense_deriv_activations<<<grid_size, block_size>>>(d_weights[layer], layer, d_current_chain_deriv);
}

__global__ void nnet_kernel_subtract_gradients(float *d_weights, float *d_weight_gradients, float *d_biases, float *d_bias_gradients, float multiplier, int rows, int cols)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < rows)
    {
        for (int j = 0; j < cols; j ++)
        {
            d_weights[i * cols + j] -= d_weight_gradients[i * cols + j] * multiplier;
            d_weight_gradients[i * cols + j] = 0;
        }
        d_biases[i] -= d_bias_gradients[i] * multiplier;
        d_bias_gradients[i] = 0;
    }
}

__host__ void nnet_subtract_gradients(float *d_weights[LAYERS], float *d_weight_gradients[LAYERS], float *d_biases[LAYERS], float *d_bias_gradients[LAYERS], float learn_rate, int batch_size)
{
    int layer;
    float mult = learn_rate / batch_size;

    dim3 grid_size((MAX_LAYER_SIZE + BLOCK_SIZE - 1)/BLOCK_SIZE);
    dim3 block_size(BLOCK_SIZE);

    for (layer = 0; layer < LAYERS; layer++)
    {
        nnet_kernel_subtract_gradients<<<grid_size, block_size>>>(d_weights[layer], d_weight_gradients[layer], d_biases[layer], d_bias_gradients[layer], mult, npl[layer+1], npl[layer]);
    }
}

__global__ void test_kernel(float *d_a, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size)
        d_a[i] = 10.0f;
}

int nnet_optimize(NeuralNet *nnet, TrainingSet *training_set, int num_mini_batches, int epochs, float learn_rate)
{
    //nnet_print(nnet);
    printf("initializing backprop... ");
    const int examples_per_batch = num_mini_batches ? training_set->num_examples / num_mini_batches : 0;
    int epoch, batch, nthExample;
    int i, j;
    int largest_layer_size = 0;

    // allocation of host data
    float *weight_gradients[LAYERS];
    float *bias_gradients[LAYERS];
    float *activations[LAYERS];
    float *chain_rule_vector;

    // initialization of host data
    for (i = 0; i <= LAYERS; i ++)
        if (npl[i] > largest_layer_size)
            largest_layer_size = npl[i];

    for (i = 0; i < LAYERS; i++)
    {
        weight_gradients[i] = laa_allocVector(npl[i + 1] * npl[i], 0);
        bias_gradients[i] = laa_allocVector(npl[i + 1], 0);
        activations[i] = laa_allocVector(npl[i + 1], 0);
    }
    chain_rule_vector = laa_allocVector(largest_layer_size, 0);

    // allocation of device data
    float *d_weight_gradients[LAYERS];
    float *d_bias_gradients[LAYERS];

    float *d_weights[LAYERS];
    float *d_biases[LAYERS];
    float *d_activations[LAYERS]; //activations don't need to be copied from host to device because their values will be set in the first forward pass

    float *d_chain_rule_vector;

    float *d_training_inputs;
    float *d_training_outputs;

    //nnet_alloc_gpu_wba(d_weights, d_weight_gradients, d_biases, d_bias_gradients, d_activations);
    //nnet_alloc_gpu_data(d_training_inputs, d_training_outputs, training_set->num_examples);

    for (i = 0; i < LAYERS; i ++)
    {
        int layer_size = sizeof(float) * npl[i+1];
        cudaErrorCheck(hipMalloc((void **)&d_weights[i], layer_size * npl[i]));
        cudaErrorCheck(hipMalloc((void **)&d_weight_gradients[i], layer_size * npl[i]));
        cudaErrorCheck(hipMalloc((void **)&d_biases[i], layer_size));
        cudaErrorCheck(hipMalloc((void **)&d_bias_gradients[i], layer_size));
        cudaErrorCheck(hipMalloc((void **)&d_activations[i], layer_size));
    }

    cudaErrorCheck(hipMalloc((void **)&d_training_inputs, INPUT_LAYER_SIZE * training_set->num_examples * sizeof(float)));
    cudaErrorCheck(hipMalloc((void **)&d_training_outputs, OUTPUT_LAYER_SIZE * training_set->num_examples * sizeof(float)));

    hipMalloc((void **)&d_chain_rule_vector, largest_layer_size * sizeof(float));

    // initialization of device data (copying from host)
    for (i = 0; i < LAYERS; i++)
    {
        // Copy the weight and bias gradients from host to device
        hipMemcpy(d_weight_gradients[i], weight_gradients[i], npl[i] * npl[i+1] * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_bias_gradients[i], bias_gradients[i], npl[i+1] * sizeof(float), hipMemcpyHostToDevice);

        // Copy the weights and biases from host neural network struct to device
        for (j = 0; j < npl[i+1]; j ++)
            cudaErrorCheck(hipMemcpy((void *)(d_weights[i] + j * npl[i]), nnet->weights[i][j], npl[i] * sizeof(float), hipMemcpyHostToDevice));
        cudaErrorCheck(hipMemcpy((void *)d_biases[i], nnet->biases[i], npl[i+1] * sizeof(float), hipMemcpyHostToDevice));
    }
    
    for (i = 0; i < training_set->num_examples; i++) 
    {
        // Copy the training data from host to device
        cudaErrorCheck(hipMemcpy((void *)(d_training_inputs + i * INPUT_LAYER_SIZE), training_set->inputs[i], INPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice));
        cudaErrorCheck(hipMemcpy((void *)(d_training_outputs + i * OUTPUT_LAYER_SIZE), training_set->outputs[i], OUTPUT_LAYER_SIZE * sizeof(float), hipMemcpyHostToDevice));
    }

    hipMemcpy(d_chain_rule_vector, chain_rule_vector, largest_layer_size*sizeof(float), hipMemcpyHostToDevice);

    printf("done\n");

    for (epoch = epochs; epoch--;)
    {
        printf("\rtraining... epoch %d/%d", epochs-epoch, epochs);
        //printf("\rtraining... %d/%d cost: %f\n", epochs-epoch, epochs, nnet_total_cost(nnet, training_set->inputs, training_set->outputs, training_set->num_examples));
        for (batch = 0; batch < num_mini_batches; batch++)
        {
            for (nthExample = batch * examples_per_batch; nthExample < (batch + 1) * examples_per_batch; nthExample++)
            {
                nnet_backprop(d_weights, d_biases, d_activations, d_weight_gradients, d_bias_gradients, d_chain_rule_vector, (float *)(d_training_inputs + nthExample * INPUT_LAYER_SIZE), (float *)(d_training_outputs + nthExample * OUTPUT_LAYER_SIZE));
            }
            nnet_subtract_gradients(d_weights, d_weight_gradients, d_biases, d_bias_gradients, learn_rate, examples_per_batch);
        }
        for (; nthExample < training_set->num_examples; nthExample++)
        {
            nnet_backprop(d_weights, d_biases, d_activations, d_weight_gradients, d_bias_gradients, d_chain_rule_vector, (float *)(d_training_inputs + nthExample * INPUT_LAYER_SIZE), (float *)(d_training_outputs + nthExample * OUTPUT_LAYER_SIZE));
        }
        nnet_subtract_gradients(d_weights, d_weight_gradients, d_biases, d_bias_gradients, learn_rate, examples_per_batch);
    }
    printf("done\n");

    //copy weights and biases from device back to host's neural network struct
    for (int i = 0; i < LAYERS; i++)
    {
        for (int j = 0; j < npl[i+1]; j ++)
            cudaErrorCheck(hipMemcpy(nnet->weights[i][j], (void *)(d_weights[i] + j * npl[i]), npl[i] * sizeof(float), hipMemcpyDeviceToHost));
        cudaErrorCheck(hipMemcpy(nnet->biases[i], d_biases[i], npl[i+1] * sizeof(float), hipMemcpyDeviceToHost));
    }

    // free host memory
    for (i = 0; i < LAYERS; i++)
    {
        laa_freeVector(weight_gradients[i]);
        laa_freeVector(bias_gradients[i]);
        laa_freeVector(activations[i]);
    }
    laa_freeVector(chain_rule_vector);

    // free device memory
    nnet_free_gpu_wba(d_weights, d_weight_gradients, d_biases, d_bias_gradients, d_activations);
    nnet_free_gpu_data(d_training_inputs, d_training_outputs);
    hipFree(d_chain_rule_vector);
    return 1;
}

//=============================

void nnetcpu_layer_function_dense_relu(float **weights, int rows, int columns, float *activations, float *bias, float *destination)
{
    for (int i = 0; i < rows; i++)
        destination[i] = relu(laa_dot(activations, weights[i], columns) + bias[i]);
}

float *nnetcpu_feed_forward(float *inputs, NeuralNet *nnet, float *activations[LAYERS])
{
    int i;
    //hidden layers:
    nnetcpu_layer_function_dense_relu(nnet->weights[0], npl[1], npl[0], inputs, nnet->biases[0], activations[0]);
    for (i = 1; i < LAYERS-1; i++)
        nnetcpu_layer_function_dense_relu(nnet->weights[i], npl[i + 1], npl[i], activations[i - 1], nnet->biases[i], activations[i]);

    //last/output layer:
    nnetcpu_layer_function_dense_relu(nnet->weights[i], npl[i + 1], npl[i], activations[i - 1], nnet->biases[i], activations[i]);

    return activations[LAYERS - 1];
}

float nnet_test_results(NeuralNet *nnet, TestingSet *test_set, int print_each_test, int print_results)
{
    float *activations[LAYERS];
    int i = 0, numWrong = 0;
    for (i = 0; i < LAYERS; i++)
    {
        activations[i] = laa_allocVector(npl[i + 1], 0);
    }
    for (i = 0; i < test_set->num_examples; i++)
    {
        if (laa_maxIndexValue(test_set->outputs[i], npl[LAYERS]) != laa_maxIndexValue(nnetcpu_feed_forward(test_set->inputs[i], nnet, activations), npl[LAYERS]))
            numWrong++;
    }
    if (print_results)
    {
        printf("\naccuracy: %d/%d (%.3f%%)\n", test_set->num_examples - numWrong, test_set->num_examples, 100 * (test_set->num_examples - numWrong) / (float)test_set->num_examples);
    }
    for (i = 0; i < LAYERS; i++)
    {
        laa_freeVector(activations[i]);
    }
    return (test_set->num_examples - numWrong) / (float)test_set->num_examples;
}

// --- --- --- --- --- --- --- --- File IO  --- --- --- --- --- --- --- ---

void nnet_print(NeuralNet *nnet)
{
    int i;
    for (i = 0; i < LAYERS; i++)
    {
        printf("\nweights:");
        laa_printMatrix(nnet->weights[i], npl[i + 1], npl[i]);
        printf("bias: ");
        laa_printVector(nnet->biases[i], npl[i + 1]);
    }
}

int nnet_save_to_file(NeuralNet *nnet, const char *fileName)
{
    FILE *filePointer = fopen(fileName, "wb");
    if (filePointer == NULL)
        return 0;

    int layers = LAYERS, i;
    fwrite(&layers, sizeof(int), 1, filePointer);
    fwrite(npl, sizeof(int), layers + 1, filePointer);
    for (i = 0; i < LAYERS; i++)
    {
        laa_writeMatrixBin(nnet->weights[i], npl[i + 1], npl[i], filePointer);
        laa_writeVectorBin(nnet->biases[i], npl[i + 1], filePointer);
    }
    fclose(filePointer);
    return 1;
}

int nnet_load_from_file(NeuralNet *nnet, const char *fileName)
{
    FILE *filePointer = fopen(fileName, "rb");
    if (filePointer == NULL)
    {
        return 0;
    }

    int layers, i;
    fread(&layers, sizeof(layers), 1, filePointer);
    int *npl = (int *)malloc(sizeof(int) * (layers + 1));
    fread(npl, sizeof(int), layers + 1, filePointer);

    for (i = 0; i < layers; i++)
    {
        laa_readMatrixBin(nnet->weights[i], filePointer);
        laa_readVectorBin(nnet->biases[i], filePointer);
    }
    fclose(filePointer);
    return 1;
}

//--- --- --- --- --- --- --- --- --- special math functions --- --- --- --- --- --- --- --- --- ---

